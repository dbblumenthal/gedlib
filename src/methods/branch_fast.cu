#include "hip/hip_runtime.h"
//
// Created by neon on 23-6-3.
//

#include <thrust/copy.h>
#include "../env/matrix.hpp"
#include "../env/ged_graph.hpp"

namespace ged {
    __device__ double *ptr{};
    __device__ std::size_t **device_sorted_edge_labels_g;
    __device__ std::size_t **device_sorted_edge_labels_h;
    __device__ int *sizes_of_array_g;
    __device__ int *sizes_of_array_h;
    __device__ double *dummy_row{}, *dummy_col{};
    __device__ int *g_deg_data{}, *h_deg_data{};
    __device__ double **relabeling_costs;
    __device__ unsigned **intersection_costs;
    __device__ double **results{};

    __device__ hipStream_t insertion, deletion, relabeling, intersection;

    static std::size_t omp_num_threads;

    extern "C"
    __host__ void
    prepare_cuda_env_(const ged::DMatrix &mat,
                      const std::map<GEDGraph::NodeID, std::vector<LabelID>> &eg,
                      const std::map<GEDGraph::NodeID, std::vector<LabelID>> &eh,
                      vector<int> &g_deg,
                      vector<int> &h_deg,
                      std::size_t num_threads
    ) {
        omp_num_threads = num_threads;

        // 有问题 没有初始化?
        hipMalloc((void **) &ptr, (mat.num_rows() * mat.num_rows()) * sizeof(size_t));
        hipMalloc((void **) &dummy_row, sizeof(double) * mat.num_rows());
        hipMalloc((void **) &dummy_col, sizeof(double) * mat.num_cols());
        thrust::copy(mat.data(), mat.data() + mat.num_rows() * mat.num_rows(), ptr);
        for (int i = 0; i < mat.num_rows(); i++) {
            dummy_row[i] = mat(0, i);
        }
        for (int i = 0; i < mat.num_cols(); i++) {
            dummy_col[i] = mat(i, 0);
        }
        // Tricky 有空修
        // 可以改异步io
        hipMalloc(device_sorted_edge_labels_g, eg.size() * sizeof(std::size_t));
        hipMalloc((void **) &sizes_of_array_g, eg.size() * sizeof(int));
        for (int i = 0; i < eg.size(); i++) {
            auto &p = device_sorted_edge_labels_g[i];
            //hipMalloc((void**)&p,sizeof(std::size_t) * eg.size());
            auto item = eg.find(i)->second;
            sizes_of_array_g[i] = item.size();
            thrust::copy(item.data(), item.data() + item.size(), p);

        }
        hipMalloc(device_sorted_edge_labels_h, eg.size() * sizeof(std::size_t));
        hipMalloc((void **) &sizes_of_array_h, eg.size() * sizeof(int));
        for (int i = 0; i < eh.size(); i++) {
            auto &p = device_sorted_edge_labels_h[i];
            //hipMalloc((void**)&p,sizeof(std::size_t) * eg.size());
            auto item = eh.find(i)->second;
            sizes_of_array_h[i] = item.size();
            thrust::copy(item.data(), item.data() + item.size(), p);

        }
        hipMalloc((void **) &g_deg_data, sizeof(int) * g_deg.size());
        hipMemcpy(g_deg_data, g_deg.data(), g_deg.size(), hipMemcpyHostToDevice);


        hipMalloc((void **) &h_deg_data, sizeof(int) * h_deg.size());
        hipMemcpy(h_deg_data, h_deg.data(), h_deg.size(), hipMemcpyHostToDevice);
    }

    __global__ void
    compute_deletion_cost(ged::GEDGraph::NodeID i,ged::GEDGraph::NodeID k,int sub) {
        extern __shared__ double reduction_helper[];

        auto tid = threadIdx.x;
        auto idx = device_sorted_edge_labels_h[k][tid];

        reduction_helper[tid] = dummy_row[idx];

        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                reduction_helper[tid + s] = thrust::min(reduction_helper[tid], reduction_helper[tid + s]);
            }
            __syncthreads();
        }

        if (tid == 0) {
            results[i][k] = reduction_helper[0] * sub * 0.5;
        }
    }

    __global__ void
    compute_insertion_cost(ged::GEDGraph::NodeID i,ged::GEDGraph::NodeID k,int sub) {
        extern __shared__ double reduction_helper[];

        auto tid = threadIdx.x;
        auto idx = device_sorted_edge_labels_g[i][tid];

        reduction_helper[tid] = dummy_col[idx];

        for (auto s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                reduction_helper[tid + s] = thrust::min(reduction_helper[tid], reduction_helper[tid + s]);
            }
            __syncthreads();
        }

        if (tid == 0) {
            results[i][k] = reduction_helper[0] * sub * 0.5;
        }
    }

    __global__ void
    compute_relabelling_cost(ged::GEDGraph::NodeID i, ged::GEDGraph::NodeID k) {
        extern __shared__ double reduction_helper[];

        // todo need recheck
        auto tid = threadIdx.x;
        auto bid = blockIdx.x;
        auto col_size = gridDim.x;
        auto pos = bid * blockDim.x + tid;
        double relabel_cost = 1e300;

        // Step1. filter out the nodes that need relabeled
        auto row = device_sorted_edge_labels_g[i][bid];
        auto col = device_sorted_edge_labels_h[k][tid];
        if (row != col)
            relabel_cost = ptr[row * col_size + col];

        // Step2. Reduce
        reduction_helper[pos] = relabel_cost;
        for (auto s = (gridDim.x * blockDim.x / 2); s > 0; s >>= 1) {
            if (pos < s) {
                reduction_helper[pos] = thrust::min(reduction_helper[pos], reduction_helper[s + pos]);
            }
            __syncthreads();
        }
        if (pos == 0) {
            relabeling_costs[i][k] = reduction_helper[0];
        }
    }


    __global__ void
    compute_multiset_intersection_size(ged::GEDGraph::NodeID i, ged::GEDGraph::NodeID k) {
        // Worst case: O(n)
        unsigned __shared__ offset_g, offset_h;
        unsigned size{};
        auto tid = threadIdx.x;
        auto sg = sizes_of_array_g[i];
        auto sh = sizes_of_array_h[k];
        offset_g = offset_h  = 0; // <- is this necessary ?

        while (true) {
            auto lg = device_sorted_edge_labels_g[i][tid + offset_g];
            auto lh = device_sorted_edge_labels_h[k][tid + offset_h];
            if (lg == lh) {
                atomicAdd(&size, 1);
                break;
            } else {
                if (tid == 0) {
                    if (lg > lh) {
                        offset_h++;
                    } else {
                        offset_g++;
                    }
                }
            }
            if (tid + offset_g >= sg || tid + offset_h >= sh)
                break;
            __syncthreads();
        }
        intersection_costs[i][k] = size;
    }

    __global__ void
    compute_substitution_cost_with_cuda(
    ) {
        // Collect node substitution cost.
        double cost{};

        auto i = blockIdx.x;
        auto k = threadIdx.x;
        auto g_deg = g_deg_data[i];
        auto h_deg = h_deg_data[k];
        /*auto g_num_nodes = gridDim.x;
        auto h_num_nodes = blockDim.x;
        auto sg = sizes_of_array_g[i];
        auto sh = sizes_of_array_h[k];*/

        double min_relabling_cost = relabeling_costs[i][k];
        unsigned intersection_size = intersection_costs[i][k];

        // Write Back
        if (thrust::min(h_deg, g_deg) - intersection_size > 0) {
            cost += static_cast<double>(thrust::min(g_deg, h_deg) - intersection_size) * min_relabling_cost * 0.5;
        }
        results[i][k] += cost;
    }

    extern "C"
    __host__  double *
    launch_kernel(int g_num_nodes, int h_num_nodes,vector<int>& g_degs,vector<int>& h_degs) {
        double **ret{};
        hipMalloc(results,sizeof(double)* g_num_nodes * h_num_nodes);
        hipMalloc((void**) &relabeling_costs,sizeof(double) * g_num_nodes * h_num_nodes);
        hipMalloc(intersection_costs,sizeof(unsigned) * g_num_nodes * h_num_nodes);

        hipStreamCreate(&insertion);
        hipStreamCreate(&deletion);
        hipStreamCreate(&relabeling);
        hipStreamCreate(&intersection);

#ifdef _OPENMP
        omp_set_num_threads(omp_num_threads - 1);
#pragma omp parallel for if(omp_num_threads > 1)
#endif
        for(auto i = 0;i<g_num_nodes;i++) {
            for (auto  k = 0;k<h_num_nodes;k++) {
                auto g_deg = g_degs[i];
                auto h_deg = h_degs[k];

                // 1. 核函数是否接受__device__参数
                // 2. malloc 2层指针是否可以按照矩阵形式访问

                // Compute insertion cost.
                if (g_deg < h_deg) {
                    compute_deletion_cost
                    <<<1, sizes_of_array_h[k], sizes_of_array_h[k] * sizeof(double), insertion>>>
                            (i,k,h_deg - g_deg);
                }

                // Compute deletion cost.
                if (g_deg > h_deg) {
                    compute_insertion_cost
                    <<<1, sizes_of_array_g[i], sizes_of_array_g[i] * sizeof(double), deletion>>>
                            (i,k,g_deg - h_deg);
                }

                // Compute relabeling cost.
                compute_relabelling_cost
                <<<g_num_nodes, h_num_nodes, g_num_nodes * h_num_nodes * sizeof(double), relabeling>>>
                        (i, k);

                // Compute multiset intersection size.
                compute_multiset_intersection_size<<<1, thrust::min(sizes_of_array_g[i], sizes_of_array_h[k])>>>(i, k);
            }
        }
        hipDeviceSynchronize();

        compute_substitution_cost_with_cuda<<<g_num_nodes,h_num_nodes>>>();
        hipStreamDestroy(insertion);
        hipStreamDestroy(deletion);
        hipStreamDestroy(relabeling);
        hipStreamDestroy(intersection);

        hipHostMalloc(ret,g_num_nodes * h_num_nodes * sizeof(double));
        hipMemcpy(ret,results,g_num_nodes * h_num_nodes * sizeof(double),hipMemcpyDeviceToHost);
        return nullptr;
    }
}